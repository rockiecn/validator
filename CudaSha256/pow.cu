#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include <dirent.h>
#include <ctype.h>
#include <sys/time.h>
// #include <openssl/sha.h>
#include <time.h>

__global__ void sha256_cuda(int diffcult, int index, BYTE *data, size_t len, long long *result, long long n)
{
    long long i = blockIdx.x * blockDim.x + threadIdx.x;
    long long start = index * n;
    // perform sha256 calculation here
    if (i < n)
    {
        long long offset = start + i;
        BYTE idx[8];
        idx[0] = (BYTE)(offset & 0xff);
        idx[1] = (BYTE)((offset >> 8) & 0xff);
        idx[2] = (BYTE)((offset >> 16) & 0xff);
        idx[3] = (BYTE)((offset >> 24) & 0xff);
        idx[4] = (BYTE)((offset >> 32) & 0xff);
        idx[5] = (BYTE)((offset >> 40) & 0xff);
        idx[6] = (BYTE)((offset >> 48) & 0xff);
        idx[7] = (BYTE)((offset >> 56) & 0xff);

        SHA256_CTX ctx;
        BYTE hash[32];
        sha256_init(&ctx);
        sha256_update(&ctx, data, len);
        sha256_update(&ctx, idx, 8);
        sha256_final(&ctx, hash);

        if (checkOutput(hash, diffcult) == 0)
        {
            *result = offset;
        }
    }
}

void pre_sha256()
{
    // compy symbols
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

void runSha256(int diffcult, int index, BYTE *data, size_t len, long long *result, long long n)
{
    int blockSize = 16;
    int numBlocks = (n + blockSize - 1) / blockSize;

    sha256_cuda<<<numBlocks, blockSize>>>(diffcult, index, data, len, result, n);
}

void byteToHexStr(const unsigned char *source, char *dest, int sourceLen)
{
    short i;
    unsigned char highByte, lowByte;

    for (i = 0; i < sourceLen; i++)
    {
        highByte = source[i] >> 4;
        lowByte = source[i] & 0x0f;

        highByte += 0x30;

        if (highByte > 0x39)
            dest[i * 2] = highByte + 0x07;
        else
            dest[i * 2] = highByte;

        lowByte += 0x30;
        if (lowByte > 0x39)
            dest[i * 2 + 1] = lowByte + 0x07;
        else
            dest[i * 2 + 1] = lowByte;
    }
    return;
}

void printOutput(const unsigned char *output, int len)
{
    char *outputHex = (char *)malloc((2 * len + 1) * sizeof(char));
    byteToHexStr(output, outputHex, len);
    outputHex[2 * len] = '\0';

    printf("%s\n", outputHex);
    free(outputHex);

    return;
}

// long long stringToLong(const char *arr)
// {
//     long long res = 0;
//     char sign;
//     int index = 0;

//     if (arr[index] == '-' || arr[index] == '+')
//     {
//         sign = arr[index++];
//     }

//     char c = arr[index++];
//     while (isdigit(c))
//     {
//         res = res * 10 + (c - '0');
//         c = arr[index++];
//     }

//     if (sign == '-')
//     {
//         return -res;
//     }

//     return res;
// }

extern "C"
{
    int generatePOW(char *rand, int len, int diffcult, long long *index)
    {
        BYTE **inputs;
        // BYTE **outputs;
        long long **indexes;
        long long count = 2 << (diffcult + 1);

        int deviceCount;
        hipGetDeviceCount(&deviceCount);

        inputs = (BYTE **)malloc(deviceCount * sizeof(BYTE *));
        // outputs = (BYTE **)malloc(deviceCount * sizeof(BYTE *));
        indexes = (long long **)malloc(deviceCount * sizeof(long long *));

        for (int i = 0; i < deviceCount; i++)
        {
            checkCudaErrors(hipSetDevice(i));

            checkCudaErrors(hipMallocManaged(&inputs[i], len * sizeof(BYTE)));
            // checkCudaErrors(hipMallocManaged(&outputs[i], 32 * sizeof(BYTE)));
            checkCudaErrors(hipMallocManaged(&indexes[i], sizeof(long long)));
            checkCudaErrors(hipMemcpy(inputs[i], rand, len, hipMemcpyHostToDevice));
        }

        for (int i = 0; i < deviceCount; i++)
        {
            checkCudaErrors(hipSetDevice(i));

            *indexes[i] = -1;
            pre_sha256();

            runSha256(diffcult, i, inputs[i], len, indexes[i], count / deviceCount);
        }

        for (int i = 0; i < deviceCount; i++)
        {
            checkCudaErrors(hipSetDevice(i));

            hipDeviceSynchronize();
        }

        // long long proof;
        for (int i = 0; i < deviceCount; i++)
            if (*indexes[i] != -1)
            {
                *index = *indexes[i];
                break;
            }

        for (int i = 0; i < deviceCount; i++)
        {
            checkCudaErrors(hipSetDevice(i));

            hipFree(inputs[i]);
            // hipFree(outputs[i]);
            hipFree(indexes[i]);
        }
        free(inputs);
        // free(outputs);
        free(indexes);

        return 0;
    }
}